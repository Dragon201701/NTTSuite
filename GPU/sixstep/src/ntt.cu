#include "hip/hip_runtime.h"
#include "../include/ntt.h"
#include "../include/utils.h"
#include <inttypes.h>
using namespace std;

void cpyVec(DATA_TYPE* src, DATA_TYPE*dst, int length){
    for(int i=0; i<length; i++){
        dst[i] = src[i];
    }
}
__device__ void modulo_cu(int64_t base, int64_t m, DATA_TYPE &a){
	int64_t result = base % m;
	a = result >= 0? result : result + m;
}

__device__ void modExp_cu(DATA_TYPE base, unsigned exp, DATA_TYPE m, DATA_TYPE &a){

	DATA_TYPE result = 1;
	
	while(exp > 0){

		if(exp % 2){

			modulo_cu(result*base, m, result);

		}

		exp = exp >> 1;
		modulo_cu(base*base,m,base);
	}

    a = result;
}


__global__ void butter_prec(DATA_TYPE *vec, DATA_TYPE *twiddle, DATA_TYPE m, unsigned shift_idx, unsigned gp, unsigned op, unsigned point) {
    //const int mid = VECTOR_SIZE >> 1;
    int p = blockIdx.x;
    int j = threadIdx.x;
    //if(j < (VECTOR_SIZE_H >> 1)){
        unsigned gg = j >> shift_idx;
        unsigned kk = j - (gg << shift_idx);
        unsigned idx1= point * gg + kk;
        unsigned idx2= idx1 + op;
        DATA_TYPE tf = twiddle[gp * kk];

        DATA_TYPE f1 = vec[p * VECTOR_SIZE_H + idx1];
        DATA_TYPE f2 = vec[p * VECTOR_SIZE_H + idx2];
        modulo_cu(f1  + f2, m, vec[p * VECTOR_SIZE_H + idx1]);

        DATA_TYPE tmp;
        modulo_cu(f1 - f2, m, tmp);

        modulo_cu(tmp * tf, m, vec[p * VECTOR_SIZE_H + idx2]);

    //}
}

__global__ void transpose(DATA_TYPE *x){
    int j = threadIdx.x;
    int i = blockIdx.x;
    //if((i < VECTOR_SIZE_H) && (j < VECTOR_SIZE_H)){
    if(i < j){
        DATA_TYPE tmp = x[i * VECTOR_SIZE_H + j];
        x[i * VECTOR_SIZE_H + j] = x[j * VECTOR_SIZE_H + i];
        x[j * VECTOR_SIZE_H + i] = tmp;
    }
    //}
}
__global__ void time_factor(DATA_TYPE *result, DATA_TYPE *vec, DATA_TYPE *twiddle, DATA_TYPE m){
    int k = threadIdx.x;
    int p = blockIdx.x;
    if(k >= p){
        DATA_TYPE wkp = twiddle[p * k]; 
        DATA_TYPE a, b;
        modulo_cu(vec[p * VECTOR_SIZE_H + k] * wkp, m, a);
        modulo_cu(vec[k * VECTOR_SIZE_H + p] * wkp, m, b);
        result[p * VECTOR_SIZE_H + k] = b;
        result[k * VECTOR_SIZE_H + p] = a;
    }
}

__global__ void  bit_reverse_cu(DATA_TYPE *result, DATA_TYPE *vec, unsigned *revArr){
    int j = threadIdx.x;
    int i = blockIdx.x;
    result[i * VECTOR_SIZE_H + revArr[j]] = vec[i * VECTOR_SIZE_H + j];
}

// GPU Mem pointer
DATA_TYPE* d_x ;
DATA_TYPE* d_y ;
DATA_TYPE* d_twiddle ;
DATA_TYPE* d_twiddle_h ;
// CPU Mem pointer
DATA_TYPE *result ;
unsigned *revArr;

void getRevArr(unsigned *revArr, unsigned n){

	unsigned num_bits = log2(n);

	unsigned reverse_num;
	for(unsigned i = 0; i < n; i++){

		reverse_num = 0;
		for(unsigned j = 0; j < num_bits; j++){

			reverse_num = reverse_num << 1;
			if(i & (1 << j)){
				reverse_num = reverse_num | 1;
			}
		}

        revArr[i] = reverse_num;
	}
}

void cudaPrepare(int n, DATA_TYPE p, DATA_TYPE g){

    hipDeviceProp_t myCuda;
    hipGetDeviceProperties(&myCuda, 0);
    cout<<"Cuda Name : "<<myCuda.name<<endl;
    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    cout<<"Cuda FreeMemory is" << freeMem<<" ; Total Memory is "<<totalMem<<endl;

    DATA_TYPE * twiddle = (DATA_TYPE*)malloc(VECTOR_SIZE * sizeof(DATA_TYPE));
    DATA_TYPE * tw      = (DATA_TYPE*)malloc(VECTOR_SIZE_H * sizeof(DATA_TYPE));

    DATA_TYPE w0 = modExp(g, (p - 1) / VECTOR_SIZE, p);

    DATA_TYPE witer = 1;
    for(int i = 0; i < VECTOR_SIZE; i++){
        twiddle[i] = witer;
        witer = modulo(witer * w0, p);
    }
    w0 = modExp(g, (p - 1) / VECTOR_SIZE_H, p);
    witer = 1;
    for(int i = 0; i < VECTOR_SIZE_H; i++){
        tw[i] = witer;
        witer = modulo(witer * w0, p);
    }


    unsigned *revArrCpu = (unsigned*)malloc(VECTOR_SIZE_H * sizeof(unsigned));
    getRevArr(revArrCpu, VECTOR_SIZE_H);

    hipMalloc(&d_x, VECTOR_SIZE * sizeof(DATA_TYPE ));
    hipMalloc(&d_y, VECTOR_SIZE * sizeof(DATA_TYPE ));
    hipMalloc(&revArr, VECTOR_SIZE_H * sizeof(DATA_TYPE ));
    hipMalloc(&d_twiddle, VECTOR_SIZE * sizeof(DATA_TYPE ));
    hipMalloc(&d_twiddle_h, VECTOR_SIZE_H * sizeof(DATA_TYPE ));

    hipMemcpy(revArr, revArrCpu, VECTOR_SIZE_H * sizeof(DATA_TYPE ), hipMemcpyHostToDevice);
    hipMemcpy(d_twiddle, twiddle, VECTOR_SIZE * sizeof(DATA_TYPE ), hipMemcpyHostToDevice);
    hipMemcpy(d_twiddle_h, tw, VECTOR_SIZE_H * sizeof(DATA_TYPE ), hipMemcpyHostToDevice);

	result = (DATA_TYPE*) malloc((int)n*sizeof(DATA_TYPE));

    free(revArrCpu);
}

void hipFree(){
    hipFree(&d_twiddle);
    free(result);
    hipFree(&d_x);
    hipFree(&d_y);
    hipFree(&revArr);
}


DATA_TYPE* six_step(DATA_TYPE * vec, int n, DATA_TYPE m, DATA_TYPE g){

    // Transfer data from cpu to gpu
    hipMemcpy(d_x, vec, VECTOR_SIZE * sizeof(DATA_TYPE ), hipMemcpyHostToDevice);

    int threadsPerBlock = VECTOR_SIZE_H;
    int blocksPerGrid   = VECTOR_SIZE_H;
    transpose<<<blocksPerGrid, threadsPerBlock>>>(d_x);
    
    for(unsigned i = 1; i <= VECTOR_ADDR_BIT_H; i++){
            unsigned shift_idx = VECTOR_ADDR_BIT_H - i;
            unsigned gp = 1 << (i - 1);
            unsigned op =  VECTOR_SIZE_H >> i;
            unsigned point = op << 1;
            butter_prec<<<blocksPerGrid, (threadsPerBlock >> 1)>>>(d_x, d_twiddle_h, m,  shift_idx, gp, op, point);
    }
    bit_reverse_cu<<<blocksPerGrid, threadsPerBlock>>>(d_y, d_x, revArr);

    time_factor<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, d_twiddle, m);

    for(unsigned i = 1; i <= VECTOR_ADDR_BIT_H; i++){
            unsigned shift_idx = VECTOR_ADDR_BIT_H - i;
            unsigned gp = 1 << (i - 1);
            unsigned op =  VECTOR_SIZE_H >> i;
            unsigned point = op << 1;
            butter_prec<<<blocksPerGrid, (threadsPerBlock >> 1)>>>(d_x, d_twiddle_h, m,  shift_idx, gp, op, point);
    }
    bit_reverse_cu<<<blocksPerGrid, threadsPerBlock>>>(d_y, d_x, revArr);

    transpose<<<blocksPerGrid, threadsPerBlock>>>(d_y);

    hipMemcpy(vec, d_y, VECTOR_SIZE * sizeof(DATA_TYPE), hipMemcpyDeviceToHost);

    return vec;
}