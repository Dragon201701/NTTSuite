#include "hip/hip_runtime.h"
#include "../include/ntt.cuh"
#include "../include/utils.h"

using namespace std;

/**
 * Perform a basic NTT on an input vector and return the result
 *
 * @param vec 	The input vector to be transformed
 * @param n	The size of the input vector
 * @param p	The prime to be used as the modulus of the transformation
 * @param r	The primitive root of the prime
 * @return 	The transformed vector
 */
DATA_TYPE *naiveNTT(DATA_TYPE *vec, unsigned n, DATA_TYPE p, DATA_TYPE r){

	DATA_TYPE k = (p - 1)/n;
	DATA_TYPE a = modExp(r,k,p);
	DATA_TYPE *result;
	result = (DATA_TYPE *) malloc(n*sizeof(DATA_TYPE));
	DATA_TYPE temp;
	for(DATA_TYPE i = 0; i < n; i++){
		temp = 0;
		for(DATA_TYPE j = 0; j < n; j++){
			temp = modulo(temp + modulo(vec[j]*modExp(a, i*j, p),p),p);
		}
		result[i] = modulo(temp,p);
	}
	return result;
}
DATA_TYPE* inPlaceNTT_DIT_precomp_golden(DATA_TYPE* vec, DATA_TYPE n, DATA_TYPE p, DATA_TYPE r, DATA_TYPE* twiddle, bool rev) {
    DATA_TYPE * result;
    result = (DATA_TYPE*)malloc(n * sizeof(DATA_TYPE));
    if (rev) {
        result = bit_reverse(vec, n);
    }
    else {
        for (DATA_TYPE i = 0; i < n; i++) {
            result[i] = vec[i];
        }
    }
    DATA_TYPE m, factor1, factor2;
    for (int i = 1; i <= log2(n); i++) {
        m = 1 << i;
        for (DATA_TYPE j = 0; j < n; j += m) {
            for (DATA_TYPE k = 0; k < m / 2; k++) {
                factor1 = result[j + k];
                factor2 = modulo(twiddle[(1 << (VECTOR_ADDR_BIT - i)) * k] * result[j + k + m / 2], p);
                result[j + k] = modulo(factor1 + factor2, p);
                result[j + k + m / 2] = modulo(factor1 - factor2, p);
            }
        }
    }
    return result;
}
int main(int argc, char **argv){
    int n = VECTOR_SIZE;
    DATA_TYPE p = (479  << 21) + 1;
  	DATA_TYPE r = 3;
    
    DATA_TYPE *vec = 0, *dev_vec = 0, *twiddle = 0, *dev_twiddle = 0;
    DATA_TYPE *result_g, *result;
    vec = randVec(VECTOR_SIZE, 1000);
    //printVec(vec, VECTOR_SIZE);
    twiddle = twiddle_cal(VECTOR_SIZE, r, p);
    //result_g = naiveNTT(vec, VECTOR_SIZE, p, r);
    hipError_t cudaStatus;
    dim3 dimGrid, dimBlock;
    cudaStatus = hipSetDevice(0);
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);
    cout<<"Cuda Name : "<<properties.name<<endl;
    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    cout<<"Cuda FreeMemory is " << freeMem<<"; Total Memory is "<<totalMem<<endl;
    cout << "using " << properties.multiProcessorCount << " multiprocessors" << endl;
    cout << "max threads per processor: " << properties.maxThreadsPerMultiProcessor << endl;

    cudaStatus = hipMalloc((void**)&dev_vec, VECTOR_SIZE * sizeof(DATA_TYPE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vec hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_vec, vec, VECTOR_SIZE * sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vec hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_twiddle, VECTOR_SIZE * sizeof(DATA_TYPE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "twiddle hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_twiddle, twiddle, VECTOR_SIZE * sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "twiddle hipMemcpy failed!");
        goto Error;
    }
    hipEvent_t kernel_start, kernel_stop, cuda_start, cuda_stop;
    hipEventCreate(&kernel_start);
    hipEventCreate(&kernel_stop);
    hipEventRecord(kernel_start,0);
    for (int i = 1; i <= log2(n); i++) {
        int m           = 1 << i;
        int maxBlocks   = m >> 1;
        int maxThreads  = 1 << (VECTOR_ADDR_BIT - i - 1);

        for(unsigned block_offset = 0; block_offset < m/2; block_offset += numberblock){
			for(unsigned  thread_offset = 0; thread_offset < VECTOR_SIZE; thread_offset += numberthread * m){
                inplaceNTT_DIF_precomp_stage << <numberblock, numberthread>> > (dev_vec, block_offset, thread_offset, m, p, dev_twiddle, i);
            }
        }
    }
    
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    hipEventRecord(kernel_stop,0);
    hipEventSynchronize(kernel_stop);
    float kernel_et;
    hipEventElapsedTime( &kernel_et, kernel_start, kernel_stop);
    printf("Kernel Time: %f milliseconds (ms) \n", kernel_et);
    
    // Copy output vector from GPU buffer to host memory.
    result = (DATA_TYPE *)malloc(sizeof(DATA_TYPE) * VECTOR_SIZE);
    cudaStatus = hipMemcpy(result, dev_vec, n * sizeof(DATA_TYPE), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "result hipMemcpy failed!");
        goto Error;
    }
    result_g = inPlaceNTT_DIT_precomp_golden(vec, VECTOR_SIZE, p, r, twiddle, false);
    compVec(result, result_g, VECTOR_SIZE, true);

Error:
    hipFree(dev_vec);
    hipFree(dev_twiddle);
    return cudaStatus;
}