#include "hip/hip_runtime.h"
#include "../include/ntt.h"
#include "../include/utils.h"
#include <inttypes.h>
using namespace std;

void cpyVec(DATA_TYPE* src, DATA_TYPE*dst, int length){
    for(int i=0; i<length; i++){
        dst[i] = src[i];
    }
}
__device__ void modulo_cu(int64_t base, int64_t m, DATA_TYPE &a){
	int64_t result = base % m;
	a = result >= 0? result : result + m;
}

__device__ void modExp_cu(DATA_TYPE base, unsigned exp, DATA_TYPE m, DATA_TYPE &a){

	DATA_TYPE result = 1;
	
	while(exp > 0){

		if(exp % 2){

			modulo_cu(result*base, m, result);

		}

		exp = exp >> 1;
		modulo_cu(base*base,m,base);
	}

    a = result;
}

__global__ void butter_prec(DATA_TYPE *yt, DATA_TYPE *xt, DATA_TYPE *twiddle, unsigned base, DATA_TYPE p) {
    int blck=   blockIdx.x;
    int r   =   threadIdx.x;
    int mid =   VECTOR_SIZE_H >> 1;
    unsigned base_addr =  blck * VECTOR_SIZE_H;
    DATA_TYPE f1 = xt[base_addr + (r << 1)];
    DATA_TYPE f2 ;
    modulo_cu(twiddle[r & base] * xt[base_addr + (r << 1) + 1], p, f2);
    modulo_cu(f1 + f2, p, yt[base_addr + r]);
    modulo_cu(f1 - f2, p, yt[base_addr + r + mid]);
}


__global__ void transpose(DATA_TYPE *x){
    int j = threadIdx.x;
    int i = blockIdx.x;
    //if((i < VECTOR_SIZE_H) && (j < VECTOR_SIZE_H)){
    if(i < j){
        DATA_TYPE tmp = x[i * VECTOR_SIZE_H + j];
        x[i * VECTOR_SIZE_H + j] = x[j * VECTOR_SIZE_H + i];
        x[j * VECTOR_SIZE_H + i] = tmp;
    }
    //}
}
__global__ void time_factor(DATA_TYPE *result, DATA_TYPE *vec, DATA_TYPE *twiddle, DATA_TYPE m){
    int k = threadIdx.x;
    int p = blockIdx.x;
    if(k >= p){
        DATA_TYPE wkp = twiddle[p * k]; 
        DATA_TYPE a, b;
        modulo_cu(vec[p * VECTOR_SIZE_H + k] * wkp, m, a);
        modulo_cu(vec[k * VECTOR_SIZE_H + p] * wkp, m, b);
        result[p * VECTOR_SIZE_H + k] = b;
        result[k * VECTOR_SIZE_H + p] = a;
    }
}

__global__ void  bit_reverse_cu(DATA_TYPE *result, DATA_TYPE *vec, unsigned *revArr){
    int j = threadIdx.x;
    int i = blockIdx.x;
    result[i * VECTOR_SIZE_H + revArr[j]] = vec[i * VECTOR_SIZE_H + j];
}

// GPU Mem pointer
DATA_TYPE* d_x ;
DATA_TYPE* d_y ;
DATA_TYPE* d_twiddle ;
DATA_TYPE* d_twiddle_h ;
// CPU Mem pointer
DATA_TYPE *result ;
unsigned *revArr;

void getRevArr(unsigned *revArr, unsigned n){

	unsigned num_bits = log2(n);

	unsigned reverse_num;
	for(unsigned i = 0; i < n; i++){

		reverse_num = 0;
		for(unsigned j = 0; j < num_bits; j++){

			reverse_num = reverse_num << 1;
			if(i & (1 << j)){
				reverse_num = reverse_num | 1;
			}
		}

        revArr[i] = reverse_num;
	}
}

void cudaPrepare(int n, DATA_TYPE p, DATA_TYPE g){

    hipDeviceProp_t myCuda;
    hipGetDeviceProperties(&myCuda, 0);
    cout<<"Cuda Name : "<<myCuda.name<<endl;
    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    cout<<"Cuda FreeMemory is" << freeMem<<" ; Total Memory is "<<totalMem<<endl;

    DATA_TYPE * twiddle = (DATA_TYPE*)malloc(VECTOR_SIZE * sizeof(DATA_TYPE));
    DATA_TYPE * tw      = (DATA_TYPE*)malloc(VECTOR_SIZE_H * sizeof(DATA_TYPE));

    DATA_TYPE w0 = modExp(g, (p - 1) / VECTOR_SIZE, p);

    DATA_TYPE witer = 1;
    for(int i = 0; i < VECTOR_SIZE; i++){
        twiddle[i] = witer;
        witer = modulo(witer * w0, p);
    }
    w0 = modExp(g, (p - 1) / VECTOR_SIZE_H, p);
    witer = 1;
    for(int i = 0; i < VECTOR_SIZE_H; i++){
        tw[i] = witer;
        witer = modulo(witer * w0, p);
    }


    unsigned *revArrCpu = (unsigned*)malloc(VECTOR_SIZE_H * sizeof(unsigned));
    getRevArr(revArrCpu, VECTOR_SIZE_H);

    hipMalloc(&d_x, VECTOR_SIZE * sizeof(DATA_TYPE ));
    hipMalloc(&d_y, VECTOR_SIZE * sizeof(DATA_TYPE ));
    hipMalloc(&revArr, VECTOR_SIZE_H * sizeof(DATA_TYPE ));
    hipMalloc(&d_twiddle, VECTOR_SIZE * sizeof(DATA_TYPE ));
    hipMalloc(&d_twiddle_h, VECTOR_SIZE_H * sizeof(DATA_TYPE ));

    hipMemcpy(revArr, revArrCpu, VECTOR_SIZE_H * sizeof(DATA_TYPE ), hipMemcpyHostToDevice);
    hipMemcpy(d_twiddle, twiddle, VECTOR_SIZE * sizeof(DATA_TYPE ), hipMemcpyHostToDevice);
    hipMemcpy(d_twiddle_h, tw, VECTOR_SIZE_H * sizeof(DATA_TYPE ), hipMemcpyHostToDevice);

	result = (DATA_TYPE*) malloc((int)n*sizeof(DATA_TYPE));

    free(revArrCpu);
}

void hipFree(){
    hipFree(&d_twiddle);
    free(result);
    hipFree(&d_x);
    hipFree(&d_y);
    hipFree(&revArr);
}


DATA_TYPE* six_step(DATA_TYPE * vec, int n, DATA_TYPE m, DATA_TYPE g){

    // Transfer data from cpu to gpu
    hipMemcpy(d_x, vec, VECTOR_SIZE * sizeof(DATA_TYPE ), hipMemcpyHostToDevice);

    int threadsPerBlock = VECTOR_SIZE_H;
    int blocksPerGrid   = VECTOR_SIZE_H;
    transpose<<<blocksPerGrid, threadsPerBlock>>>(d_x);
    
    bit_reverse_cu<<<blocksPerGrid, threadsPerBlock>>>(d_y, d_x, revArr);
    for (unsigned c = VECTOR_ADDR_BIT_H; c >= 2; c-=2){
        unsigned base = -1 << (c - 1);
        //unsigned base = -1 << (c - 1);
        butter_prec<<<blocksPerGrid, (threadsPerBlock >> 1)>>>(d_x, d_y, d_twiddle_h, base, m);

        base = -1 << (c - 2);
        butter_prec<<<blocksPerGrid, (threadsPerBlock >> 1)>>>(d_y, d_x, d_twiddle_h, base, m);
        //printVec(xt,n);
    }
    if(VECTOR_ADDR_BIT_H % 2){
        unsigned base = -1;
        butter_prec<<<blocksPerGrid, (threadsPerBlock >> 1)>>>(d_x, d_y, d_twiddle_h, base, m);
        DATA_TYPE *tmp = d_x;
        d_x = d_y;
        d_y = tmp;
    }

    time_factor<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_y, d_twiddle, m);

    bit_reverse_cu<<<blocksPerGrid, threadsPerBlock>>>(d_y, d_x, revArr);
    for (unsigned c = VECTOR_ADDR_BIT_H; c >= 2; c-=2){
        unsigned base = -1 << (c - 1);
        //unsigned base = -1 << (c - 1);
        butter_prec<<<blocksPerGrid, (threadsPerBlock >> 1)>>>(d_x, d_y, d_twiddle_h, base, m);

        base = -1 << (c - 2);
        butter_prec<<<blocksPerGrid, (threadsPerBlock >> 1)>>>(d_y, d_x, d_twiddle_h, base, m);
        //printVec(xt,n);
    }
    if(VECTOR_ADDR_BIT_H % 2){
        unsigned base = -1;
        butter_prec<<<blocksPerGrid, (threadsPerBlock >> 1)>>>(d_x, d_y, d_twiddle_h, base, m);
        DATA_TYPE *tmp = d_x;
        d_x = d_y;
        d_y = tmp;
    }

    transpose<<<blocksPerGrid, threadsPerBlock>>>(d_y);

    hipMemcpy(vec, d_y, VECTOR_SIZE * sizeof(DATA_TYPE), hipMemcpyDeviceToHost);

    return vec;
}