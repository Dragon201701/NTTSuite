#include "hip/hip_runtime.h"
#include "../include/ntt.cuh"
#include "../include/utils.h"

using namespace std;

/**
 * Perform a basic NTT on an input vector and return the result
 *
 * @param vec 	The input vector to be transformed
 * @param n	The size of the input vector
 * @param p	The prime to be used as the modulus of the transformation
 * @param r	The primitive root of the prime
 * @return 	The transformed vector
 */
DATA_TYPE *naiveNTT(DATA_TYPE *vec, unsigned n, DATA_TYPE p, DATA_TYPE r){

	DATA_TYPE k = (p - 1)/n;
	DATA_TYPE a = modExp(r,k,p);
	DATA_TYPE *result;
	result = (DATA_TYPE *) malloc(n*sizeof(DATA_TYPE));
	DATA_TYPE temp;
	for(DATA_TYPE i = 0; i < n; i++){
		temp = 0;
		for(DATA_TYPE j = 0; j < n; j++){
			temp = modulo(temp + modulo(vec[j]*modExp(a, i*j, p),p),p);
		}
		result[i] = modulo(temp,p);
	}
	return result;
}
DATA_TYPE* inPlaceNTT_DIF_precomp_golden(DATA_TYPE* vec, DATA_TYPE n, DATA_TYPE p, DATA_TYPE r, DATA_TYPE* twiddle, bool rev) {
    DATA_TYPE * result;
    result = (DATA_TYPE*)malloc(n * sizeof(DATA_TYPE));
    DATA_TYPE m;
    for (int i = log2(n); i >= 1; i--) {
        m = 1 << i;
        for (DATA_TYPE j = 0; j < n; j += m) {
            for (DATA_TYPE k = 0; k < m / 2; k++) {
    	        DATA_TYPE w = twiddle[(1 << (VECTOR_ADDR_BIT - i)) * k];
	            DATA_TYPE f1 = vec[j + k];
	            DATA_TYPE f2 = vec[j + k + m/2];
	            vec[j + k] = modulo(f1 + f2, p);
	            vec[j + k + m/2] = modulo(w * modulo(f1 - f2, p), p);
            }
        }
    }
    if (rev) {
        result = bit_reverse(vec, n);
    }
    else {
        for (DATA_TYPE i = 0; i < n; i++) {
            result[i] = vec[i];
        }
    }
    return result;
}

float nttcuda(int n, DATA_TYPE p, DATA_TYPE r){
    DATA_TYPE *vec, *twiddle;
    DATA_TYPE *result_g, *result;

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);
    cout<<"Cuda Name : "<<properties.name<<endl;
    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    cout<<"Cuda FreeMemory is " << freeMem<<"; Total Memory is "<<totalMem<<endl;
    cout << "using " << properties.multiProcessorCount << " multiprocessors" << endl;
    cout << "max threads per processor: " << properties.maxThreadsPerMultiProcessor << endl;

    int threadsPerBlock = 1024;
    int blocksPerGrid = ((VECTOR_SIZE>>1) + threadsPerBlock - 1) / threadsPerBlock;

    int threadsPerBlockRev = 512;
    int blocksPerGridRev = (VECTOR_SIZE + threadsPerBlock - 1) / threadsPerBlock;

    unsigned t = (unsigned)log2(n);

    hipEvent_t malloc_start, malloc_stop, cuda_start, cuda_stop;
    hipEventCreate(&malloc_start);
    hipEventCreate(&malloc_stop);
    hipEventCreate(&cuda_start);
    hipEventCreate(&cuda_stop);

    hipEventRecord(malloc_start,0);
    cudaStatus = hipMallocManaged(&vec, VECTOR_SIZE * sizeof(DATA_TYPE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vec hipMallocManaged failed!");
        goto Error;
    }
    cudaStatus = hipMallocManaged(&result, VECTOR_SIZE * sizeof(DATA_TYPE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vec hipMallocManaged failed!");
        goto Error;
    }
    
    cudaStatus = hipMallocManaged(&twiddle, VECTOR_SIZE * sizeof(DATA_TYPE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "twiddle hipMallocManaged failed!");
        goto Error;
    }
    hipEventRecord(malloc_stop,0);
    hipEventSynchronize(malloc_stop);
    float malloc_et;
    hipEventElapsedTime( &malloc_et, malloc_start, malloc_stop);
    //printf("CUDA Time: %f milliseconds (ms) \n", cuda_et);

    randVec(VECTOR_SIZE, vec, 1000);
    result_g = naiveNTT(vec, VECTOR_SIZE, p, r);
    twiddle_cal(VECTOR_SIZE, r, p, twiddle);



    hipEventRecord(cuda_start,0);
    // hipEventRecord(kernel_start,0);

    for (unsigned c = t; c >= 1; c--){
        unsigned base = -1 << (c - 1);
        //unsigned base = -1 << (c - 1);
        butter_prec<<<blocksPerGrid, threadsPerBlock>>>(result, vec, twiddle, base, p);
        hipMemcpy(vec, result, VECTOR_SIZE * sizeof(DATA_TYPE), hipMemcpyDeviceToDevice);
        //printVec(xt,n);
    }
    
    hipEventRecord(cuda_stop,0);
    hipEventSynchronize(cuda_stop);
    float cuda_et;
    hipEventElapsedTime( &cuda_et, cuda_start, cuda_stop);
    printf("CUDA Time: %f milliseconds (ms) \n", cuda_et + malloc_et);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    compVec(result, result_g, VECTOR_SIZE, true);
    hipFree(vec);
    hipFree(result);
    hipFree(twiddle);
    return cuda_et + malloc_et;
Error:
    hipFree(vec);
    hipFree(twiddle);
    return -1;

}
int main(int argc, char **argv){
    int n = VECTOR_SIZE;
    DATA_TYPE p = (479  << 21) + 1;
  	DATA_TYPE r = 3;
    
    float result = 0;

    for (int i = 0; i < 100; i++){
        result += nttcuda(n, p, r);
    }
    cout << "Average for 100 tests: " << result/100 << endl;
    
}