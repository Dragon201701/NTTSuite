#include "hip/hip_runtime.h"
#include "../include/ntt.cuh"

using namespace std;

__device__ DATA_TYPE modulo_dev(DATA_TYPE_SIGNED base, DATA_TYPE_SIGNED m) {
	DATA_TYPE_SIGNED result = base % m;
	return (result >= 0) ? result : result + m;
}

__device__ void modulo_cu(int64_t base, int64_t m, DATA_TYPE &a){
	int64_t result = base % m;
	a = result >= 0? result : result + m;
}

__device__ DATA_TYPE modExp_cu(DATA_TYPE base, unsigned exp, DATA_TYPE m){
	DATA_TYPE result = 1;
    while(exp > 0){
        result = modulo_dev(result * base, m);
        exp--;
    }
    return result;
}
__global__ void inplaceNTT_DIF_precomp_stage(DATA_TYPE* vec, int block_offset, int thread_offset,
		DATA_TYPE m, DATA_TYPE p, DATA_TYPE* twiddle, int i) {
	int k = blockIdx.x  + block_offset;
	int j = m * threadIdx.x + thread_offset;
	if((k >= (m >> 1)) || (j >= VECTOR_SIZE))
		return;
    	DATA_TYPE w = twiddle[(1 << (VECTOR_ADDR_BIT - i)) * k];
	// DATA_TYPE f1 = vec[j + k];
	// DATA_TYPE f2 = modulo_dev(w * vec[j + k + m / 2], p);
	// vec[j + k] = modulo_dev(f1 + f2, p);
	// vec[j + k + m / 2] = modulo_dev(f1 - f2, p);
	DATA_TYPE f1 = vec[j + k];
	DATA_TYPE f2 = vec[j + k + m/2];
	vec[j + k] = modulo_dev(f1 + f2, p);
	vec[j + k + m/2] = modulo_dev(w * modulo_dev(f1 - f2, p), p);
}
__global__ void  bit_reverse_cu(DATA_TYPE *result, DATA_TYPE *vec, unsigned num_bits){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < VECTOR_SIZE){
		unsigned reverse_num = 0;
		for(unsigned j = 0; j < num_bits; j++){
			reverse_num = reverse_num << 1;
			if(i & (1 << j)){
				reverse_num = reverse_num | 1;
			}
		}
		result[reverse_num] = vec[i];
    }
}

__global__ void butter_prec(DATA_TYPE *yt, DATA_TYPE *xt, DATA_TYPE *twiddle, unsigned base, DATA_TYPE p) {
    int r = blockIdx.x * blockDim.x + threadIdx.x;
    int mid = VECTOR_SIZE >> 1;
    if(r < mid){
        DATA_TYPE f1 = xt[r<<1];
        DATA_TYPE f2 ;
        modulo_cu(twiddle[r & base]* xt[(r<<1) + 1], p, f2);
        modulo_cu(f1 + f2, p, yt[r]);
        modulo_cu(f1 - f2, p, yt[r + mid]);
    }
}