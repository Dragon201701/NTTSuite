#include "hip/hip_runtime.h"
#include "../include/ntt.cuh"
#include "../include/utils.h"

using namespace std;

/**
 * Perform a basic NTT on an input vector and return the result
 *
 * @param vec 	The input vector to be transformed
 * @param n	The size of the input vector
 * @param p	The prime to be used as the modulus of the transformation
 * @param r	The primitive root of the prime
 * @return 	The transformed vector
 */
DATA_TYPE *naiveNTT(DATA_TYPE *vec, unsigned n, DATA_TYPE p, DATA_TYPE r){

	DATA_TYPE k = (p - 1)/n;
	DATA_TYPE a = modExp(r,k,p);
	DATA_TYPE *result;
	result = (DATA_TYPE *) malloc(n*sizeof(DATA_TYPE));
	DATA_TYPE temp;
	for(DATA_TYPE i = 0; i < n; i++){
		temp = 0;
		for(DATA_TYPE j = 0; j < n; j++){
			temp = modulo(temp + modulo(vec[j]*modExp(a, i*j, p),p),p);
		}
		result[i] = modulo(temp,p);
	}
	return result;
}
DATA_TYPE* inPlaceNTT_DIF_precomp_golden(DATA_TYPE* vec, DATA_TYPE n, DATA_TYPE p, DATA_TYPE r, DATA_TYPE* twiddle, bool rev) {
    DATA_TYPE * result;
    result = (DATA_TYPE*)malloc(n * sizeof(DATA_TYPE));
    DATA_TYPE m;
    for (int i = log2(n); i >= 1; i--) {
        m = 1 << i;
        for (DATA_TYPE j = 0; j < n; j += m) {
            for (DATA_TYPE k = 0; k < m / 2; k++) {
    	        DATA_TYPE w = twiddle[(1 << (VECTOR_ADDR_BIT - i)) * k];
	            DATA_TYPE f1 = vec[j + k];
	            DATA_TYPE f2 = vec[j + k + m/2];
	            vec[j + k] = modulo(f1 + f2, p);
	            vec[j + k + m/2] = modulo(w * modulo(f1 - f2, p), p);
            }
        }
    }
    if (rev) {
        result = bit_reverse(vec, n);
    }
    else {
        for (DATA_TYPE i = 0; i < n; i++) {
            result[i] = vec[i];
        }
    }
    return result;
}

float nttcuda(int n, DATA_TYPE p, DATA_TYPE r){
    DATA_TYPE *vec, *twiddle;
    DATA_TYPE *result_g, *result;

    hipError_t cudaStatus;
    cudaStatus = hipSetDevice(0);
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);
    cout<<"Cuda Name : "<<properties.name<<endl;
    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    cout<<"Cuda FreeMemory is " << freeMem<<"; Total Memory is "<<totalMem<<endl;
    cout << "using " << properties.multiProcessorCount << " multiprocessors" << endl;
    cout << "max threads per processor: " << properties.maxThreadsPerMultiProcessor << endl;

    hipEvent_t malloc_start, malloc_stop, cuda_start, cuda_stop;
    hipEventCreate(&malloc_start);
    hipEventCreate(&malloc_stop);
    hipEventCreate(&cuda_start);
    hipEventCreate(&cuda_stop);

    hipEventRecord(malloc_start,0);
    cudaStatus = hipMallocManaged(&vec, VECTOR_SIZE * sizeof(DATA_TYPE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vec hipMallocManaged failed!");
        goto Error;
    }
    
    cudaStatus = hipMallocManaged(&twiddle, VECTOR_SIZE * sizeof(DATA_TYPE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "twiddle hipMallocManaged failed!");
        goto Error;
    }
    hipEventRecord(malloc_stop,0);
    hipEventSynchronize(malloc_stop);
    float malloc_et;
    hipEventElapsedTime( &malloc_et, malloc_start, malloc_stop);
    //printf("CUDA Time: %f milliseconds (ms) \n", cuda_et);

    randVec(VECTOR_SIZE, vec, 1000);
    result_g = naiveNTT(vec, VECTOR_SIZE, p, r);
    twiddle_cal(VECTOR_SIZE, r, p, twiddle);

    hipEventRecord(cuda_start,0);
    // hipEventRecord(kernel_start,0);
    for (int i = log2(n); i >= 1; i--) {
        int m           = 1 << i;
        int maxBlocks   = m >> 1;
        int maxThreads  = 1 << (VECTOR_ADDR_BIT - i - 1);

        for(unsigned block_offset = 0; block_offset < m/2; block_offset += numberblock){
			for(unsigned  thread_offset = 0; thread_offset < VECTOR_SIZE; thread_offset += numberthread * m){
                inplaceNTT_DIF_precomp_stage << <numberblock, numberthread>> > (vec, block_offset, thread_offset, m, p, twiddle, i);
            }
        }
    }
    
    hipEventRecord(cuda_stop,0);
    hipEventSynchronize(cuda_stop);
    float cuda_et;
    hipEventElapsedTime( &cuda_et, cuda_start, cuda_stop);
    printf("CUDA Time: %f milliseconds (ms) \n", cuda_et + malloc_et);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    compVec(vec, result_g, VECTOR_SIZE, true);

    return cuda_et + malloc_et;
Error:
    hipFree(vec);
    hipFree(twiddle);
    return -1;

}
int main(int argc, char **argv){
    int n = VECTOR_SIZE;
    DATA_TYPE p = (479  << 21) + 1;
  	DATA_TYPE r = 3;
    
    float result = 0;

    for (int i = 0; i < 100; i++){
        result += nttcuda(n, p, r);
    }
    cout << "Average for 100 tests: " << result/100 << endl;
    
}