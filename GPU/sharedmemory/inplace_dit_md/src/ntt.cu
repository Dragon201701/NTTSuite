#include "hip/hip_runtime.h"
#include "../include/ntt.cuh"

using namespace std;

__device__ DATA_TYPE modulo_dev(DATA_TYPE_SIGNED base, DATA_TYPE_SIGNED m) {
	DATA_TYPE_SIGNED result = base % m;
	return (result >= 0) ? result : result + m;
}

__device__ DATA_TYPE modExp_cu(DATA_TYPE base, unsigned exp, DATA_TYPE m){
	DATA_TYPE result = 1;
    while(exp > 0){
        result = modulo_dev(result * base, m);
        exp--;
    }
    return result;
}
__global__ void inplaceNTT_DIT_precomp_stage(DATA_TYPE* vec, int block_offset, int thread_offset,
		DATA_TYPE m, DATA_TYPE p, DATA_TYPE* twiddle, int i) {
	int k = blockIdx.x  + block_offset;
	int j = m * threadIdx.x + thread_offset;
	if((k >= (m >> 1)) || (j >= VECTOR_SIZE))
		return;
    DATA_TYPE w = twiddle[(1 << (VECTOR_ADDR_BIT - i)) * k];
	DATA_TYPE f1 = vec[j + k];
	DATA_TYPE f2 = modulo_dev(w * vec[j + k + m / 2], p);
	vec[j + k] = modulo_dev(f1 + f2, p);
	vec[j + k + m / 2] = modulo_dev(f1 - f2, p);
}
__global__ void  bit_reverse_cu(DATA_TYPE *result, DATA_TYPE *vec, unsigned num_bits){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < VECTOR_SIZE){
		unsigned reverse_num = 0;
		for(unsigned j = 0; j < num_bits; j++){
			reverse_num = reverse_num << 1;
			if(i & (1 << j)){
				reverse_num = reverse_num | 1;
			}
		}
		result[reverse_num] = vec[i];
    }
}