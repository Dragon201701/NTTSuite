#include "hip/hip_runtime.h"
#include "../include/ntt.cuh"

using namespace std;

__device__ DATA_TYPE modulo_dev(DATA_TYPE_SIGNED base, DATA_TYPE_SIGNED m) {
	DATA_TYPE_SIGNED result = base % m;
	return (result >= 0) ? result : result + m;
}

__device__ DATA_TYPE modExp_cu(DATA_TYPE base, unsigned exp, DATA_TYPE m){
	DATA_TYPE result = 1;
    while(exp > 0){
        result = modulo_dev(result * base, m);
        exp--;
    }
    return result;
}
__global__ void inplaceNTT_DIT_precomp_stage(DATA_TYPE* vec, int batch,
		DATA_TYPE m, DATA_TYPE p, DATA_TYPE* twiddle, int thread_offset) {
	int j = blockIdx.x * m + batch;
	int k = threadIdx.x + thread_offset;
	DATA_TYPE f1 = vec[j + k];
	DATA_TYPE f2 = modulo_dev(twiddle[(DATA_TYPE)m/2 - 1 + k] * vec[j + k + m / 2], p);
	vec[j + k] = modulo_dev(f1 + f2, p);
	vec[j + k + m / 2] = modulo_dev(f1 - f2, p);
}
__global__ void  bit_reverse_cu(DATA_TYPE *result, DATA_TYPE *vec, unsigned num_bits){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < VECTOR_SIZE){
		unsigned reverse_num = 0;
		for(unsigned j = 0; j < num_bits; j++){
			reverse_num = reverse_num << 1;
			if(i & (1 << j)){
				reverse_num = reverse_num | 1;
			}
		}
		result[reverse_num] = vec[i];
    }
}