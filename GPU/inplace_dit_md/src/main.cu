#include "hip/hip_runtime.h"
#include "../include/ntt.cuh"
#include "../include/utils.h"

using namespace std;

/**
 * Perform a basic NTT on an input vector and return the result
 *
 * @param vec 	The input vector to be transformed
 * @param n	The size of the input vector
 * @param p	The prime to be used as the modulus of the transformation
 * @param r	The primitive root of the prime
 * @return 	The transformed vector
 */
DATA_TYPE *naiveNTT(DATA_TYPE *vec, unsigned n, DATA_TYPE p, DATA_TYPE r){

	DATA_TYPE k = (p - 1)/n;
	DATA_TYPE a = modExp(r,k,p);
	DATA_TYPE *result;
	result = (DATA_TYPE *) malloc(n*sizeof(DATA_TYPE));
	DATA_TYPE temp;
	for(DATA_TYPE i = 0; i < n; i++){
		temp = 0;
		for(DATA_TYPE j = 0; j < n; j++){
			temp = modulo(temp + modulo(vec[j]*modExp(a, i*j, p),p),p);
		}
		result[i] = modulo(temp,p);
	}
	return result;
}

int main(int argc, char **argv){
    int n = VECTOR_SIZE;
    DATA_TYPE p = (479  << 21) + 1;
  	DATA_TYPE r = 3;
    
    DATA_TYPE *vec = 0, *dev_vec = 0, *twiddle = 0, *dev_twiddle = 0;
    DATA_TYPE *result_g, *result;
    vec = randVec(VECTOR_SIZE, 1000);
    printVec(vec, VECTOR_SIZE);
    twiddle = twiddle_cal(VECTOR_SIZE, r, p);
    result_g = naiveNTT(vec, VECTOR_SIZE, p, r);

    hipError_t cudaStatus;
    dim3 dimGrid, dimBlock;
    cudaStatus = hipSetDevice(0);
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, 0);
    cout<<"Cuda Name : "<<properties.name<<endl;
    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    cout<<"Cuda FreeMemory is " << freeMem<<"; Total Memory is "<<totalMem<<endl;
    cout << "using " << properties.multiProcessorCount << " multiprocessors" << endl;
    cout << "max threads per processor: " << properties.maxThreadsPerMultiProcessor << endl;

    hipMalloc((void**)&dev_vec, VECTOR_SIZE * sizeof(DATA_TYPE));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "dev vec hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(dev_vec, vec, VECTOR_SIZE * sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "vec hipMemcpy failed!");
        goto Error;
    }

    hipMalloc((void**)&dev_twiddle, VECTOR_SIZE * sizeof(DATA_TYPE));
    cudaStatus = hipMemcpy(dev_twiddle, twiddle, VECTOR_SIZE * sizeof(DATA_TYPE), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "twiddle hipMemcpy failed!");
        goto Error;
    }
    for (int i = 1; i <= log2(n); i++) {
        uint64_t m = (int)pow(2, i);
        uint64_t k_ = (p - 1) / m;
        uint64_t a = modExp(r, k_, p);
        int numblocks = 64, maxthreads = 1024, numthreads = m / 2, thread_offset = 0;

        for (int batch = 0; batch < n; batch += m * numblocks) {
            // if (numthreads > maxthreads) {
                dimBlock.x = maxthreads;
                for (int stage = 0; stage < numthreads / maxthreads; stage++) {
                    inplaceNTT_DIT_precomp_stage << <numblocks, dimBlock >> > (dev_vec, batch, m, p, dev_twiddle, stage * maxthreads);
                }
            // }
            // else {
            //     dimBlock.x = numthreads;
            //     inplaceNTT_DIT_precomp_stage << <numblocks, dimBlock >> > (dev_vec, batch, m, p, dev_twiddle, 0);
            // }
        }
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    // Copy output vector from GPU buffer to host memory.
    result = (DATA_TYPE *)malloc(sizeof(DATA_TYPE) * VECTOR_SIZE);
    cudaStatus = hipMemcpy(result, dev_vec, n * sizeof(uint64_t), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "result hipMemcpy failed!");
        goto Error;
    }
    compVec(result_g, result, VECTOR_SIZE, true);

Error:
    hipFree(dev_vec);
    hipFree(dev_twiddle);
    return cudaStatus;
}