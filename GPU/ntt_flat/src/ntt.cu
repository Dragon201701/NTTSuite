#include "hip/hip_runtime.h"
#include "../include/ntt.h"
#include "../include/utils.h"
#include <inttypes.h>
using namespace std;
#define TIME_TEST

void cpyVec(DATA_TYPE* src, DATA_TYPE*dst, int length){
    for(int i=0; i<length; i++){
        dst[i] = src[i];
    }
}
__device__ void modulo_cu(int64_t base, int64_t m, DATA_TYPE &a){
	int64_t result = base % m;
	a = result >= 0? result : result + m;
}

__device__ void modExp_cu(DATA_TYPE base, unsigned exp, DATA_TYPE m, DATA_TYPE &a){

	DATA_TYPE result = 1;
	
	while(exp > 0){

		if(exp % 2){

			modulo_cu(result*base, m, result);

		}

		exp = exp >> 1;
		modulo_cu(base*base,m,base);
	}

    a = result;
}


__global__ void butter_prec(DATA_TYPE *vec, DATA_TYPE *twiddle, DATA_TYPE p, unsigned shift_idx, unsigned gp, unsigned op, unsigned point) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int mid = VECTOR_SIZE >> 1;
    if(j < mid){
        unsigned g = j >> shift_idx;
        unsigned k = j - (g << shift_idx);
        unsigned idx1= point * g + k;
        unsigned idx2= idx1 + op;
        DATA_TYPE tf = twiddle[gp * k];
        //cout<<idx1<<" "<<idx2<<" "<<gp*t<<endl;
        DATA_TYPE f1 = vec[idx1];
        DATA_TYPE f2 = vec[idx2];
        modulo_cu(f1 + f2, p, vec[idx1]);
        DATA_TYPE tmp;
        modulo_cu(f1 - f2, p, tmp);
        modulo_cu(tmp * tf, p, vec[idx2]);
    }
}

// GPU Mem pointer
DATA_TYPE* d_x ;
DATA_TYPE* d_y ;
DATA_TYPE* d_twiddle ;
// CPU Mem pointer
DATA_TYPE *result ;

void cudaPrepare(int n, DATA_TYPE p, DATA_TYPE g){
    hipDeviceProp_t myCuda;
    hipGetDeviceProperties(&myCuda, 0);
    cout<<"Cuda Name : "<<myCuda.name<<endl;
    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    cout<<"Cuda FreeMemory is" << freeMem<<" ; Total Memory is "<<totalMem<<endl;

    DATA_TYPE * twiddle = (DATA_TYPE*)malloc(n * sizeof(DATA_TYPE));
    DATA_TYPE w0 = modExp(g, (p - 1) / n, p);
    DATA_TYPE witer = 1;
    for(int i = 0; i < n; i++){
        twiddle[i] = witer;
        witer = modulo(witer * w0, p);
    }

    hipMalloc(&d_x, VECTOR_SIZE * sizeof(DATA_TYPE ));
    hipMalloc(&d_y, VECTOR_SIZE * sizeof(DATA_TYPE ));
    hipMalloc(&d_twiddle, VECTOR_SIZE * sizeof(DATA_TYPE ));
    hipMemcpy(d_twiddle, twiddle, VECTOR_SIZE * sizeof(DATA_TYPE ), hipMemcpyHostToDevice);
	result = (DATA_TYPE*) malloc((int)n*sizeof(DATA_TYPE));
}

void hipFree(){
    hipFree(&d_twiddle);
    free(result);
    hipFree(&d_x);
    hipFree(&d_y);
}

__global__ void  bit_reverse_cu(DATA_TYPE *result, DATA_TYPE *vec, unsigned num_bits){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < VECTOR_SIZE){
		unsigned reverse_num = 0;
		for(unsigned j = 0; j < num_bits; j++){
			reverse_num = reverse_num << 1;
			if(i & (1 << j)){
				reverse_num = reverse_num | 1;
			}
		}
		result[reverse_num] = vec[i];
    }
}

DATA_TYPE* ntt_flat(DATA_TYPE * vec,  int n, DATA_TYPE p, DATA_TYPE g){

    DATA_TYPE t = (DATA_TYPE)log2(n);

#ifndef TIME_TEST
    hipMemcpy(d_x, vec, VECTOR_SIZE * sizeof(DATA_TYPE ), hipMemcpyHostToDevice);
#endif
    int threadsPerBlock = 256;
    int blocksPerGrid = ((VECTOR_SIZE>>1) + threadsPerBlock - 1) / threadsPerBlock;

    for(unsigned i = 1; i <= t; i++){
        unsigned shift_idx = t - i;
        unsigned gp = 1 << (i - 1);
        unsigned op =  VECTOR_SIZE >> i;
        unsigned point = op << 1;
        butter_prec<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_twiddle, p,  shift_idx, gp, op, point);
    }


    int threadsPerBlockRev = 256;
    int blocksPerGridRev = (VECTOR_SIZE + threadsPerBlock - 1) / threadsPerBlock;
#ifndef TIME_TEST
    bit_reverse_cu<<<blocksPerGridRev, threadsPerBlockRev>>>(d_y, d_x, t);

    hipMemcpy(result, d_y, VECTOR_SIZE * sizeof(DATA_TYPE ), hipMemcpyDeviceToHost);
#endif

    return result;
}