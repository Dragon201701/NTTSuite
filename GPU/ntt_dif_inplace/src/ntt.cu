#include "hip/hip_runtime.h"
#include "../include/ntt.h"
#include "../include/utils.h"
#include <inttypes.h>
using namespace std;


#define TIME_TEST
void cpyVec(DATA_TYPE* src, DATA_TYPE*dst, int length){
    for(int i=0; i<length; i++){
        dst[i] = src[i];
    }
}
__device__ void modulo_cu(int64_t base, int64_t m, DATA_TYPE &a){
	int64_t result = base % m;
	a = result >= 0? result : result + m;
}

__device__ void modExp_cu(DATA_TYPE base, unsigned exp, DATA_TYPE m, DATA_TYPE &a){

	DATA_TYPE result = 1;
	
	while(exp > 0){

		if(exp % 2){

			modulo_cu(result*base, m, result);

		}

		exp = exp >> 1;
		modulo_cu(base*base,m,base);
	}

    a = result;
}


__global__ void butter_prec(DATA_TYPE *result, DATA_TYPE *twiddle, DATA_TYPE p, unsigned m, unsigned j, unsigned mid, unsigned i) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if(k >= mid)
        return ;
    DATA_TYPE w = twiddle[(1 << (VECTOR_ADDR_BIT - i)) * k];
    DATA_TYPE factor1   = result[j + k];
    DATA_TYPE factor2   = result[j + k + mid];
    modulo_cu(factor1 + factor2, p, result[j + k]);
    DATA_TYPE tmp;
    modulo_cu(factor1 - factor2, p, tmp);
    modulo_cu(tmp * w, p, result[j + k + mid]);
}

// GPU Mem pointer
DATA_TYPE* d_x ;
DATA_TYPE* d_y ;
DATA_TYPE* d_twiddle ;
// CPU Mem pointer
DATA_TYPE *result ;

void cudaPrepare(int n, DATA_TYPE p, DATA_TYPE g){
    hipDeviceProp_t myCuda;
    hipGetDeviceProperties(&myCuda, 0);
    cout<<"Cuda Name : "<<myCuda.name<<endl;
    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    cout<<"Cuda FreeMemory is" << freeMem<<" ; Total Memory is "<<totalMem<<endl;

    DATA_TYPE * twiddle = (DATA_TYPE*)malloc(n * sizeof(DATA_TYPE));
    DATA_TYPE w0 = modExp(g, (p - 1) / n, p);
    DATA_TYPE witer = 1;
    for(int i = 0; i < n; i++){
        twiddle[i] = witer;
        witer = modulo(witer * w0, p);
    }

    hipMalloc(&d_x, VECTOR_SIZE * sizeof(DATA_TYPE ));
    hipMalloc(&d_y, VECTOR_SIZE * sizeof(DATA_TYPE ));
    hipMalloc(&d_twiddle, VECTOR_SIZE * sizeof(DATA_TYPE ));
    hipMemcpy(d_twiddle, twiddle, VECTOR_SIZE * sizeof(DATA_TYPE ), hipMemcpyHostToDevice);
	result = (DATA_TYPE*) malloc((int)n*sizeof(DATA_TYPE));
}

void hipFree(){
    hipFree(&d_twiddle);
    free(result);
    hipFree(&d_x);
    hipFree(&d_y);
}

__global__ void  bit_reverse_cu(DATA_TYPE *result, DATA_TYPE *vec, unsigned num_bits){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < VECTOR_SIZE){
		unsigned reverse_num = 0;
		for(unsigned j = 0; j < num_bits; j++){
			reverse_num = reverse_num << 1;
			if(i & (1 << j)){
				reverse_num = reverse_num | 1;
			}
		}
		result[reverse_num] = vec[i];
    }
}

DATA_TYPE* ntt_flat(DATA_TYPE * vec,  int n, DATA_TYPE p, DATA_TYPE g){

    DATA_TYPE t = (DATA_TYPE)log2(n);
#ifndef TIME_TEST
    hipMemcpy(d_x, vec, VECTOR_SIZE * sizeof(DATA_TYPE ), hipMemcpyHostToDevice);
#endif

    int threadsPerBlock = 256;
    int blocksPerGrid = ((VECTOR_SIZE>>1) + threadsPerBlock - 1) / threadsPerBlock;

	for(unsigned i = t; i >= 1; i--){ 
		unsigned m = 1 << i;
		unsigned mid = m >> 1;
		for(unsigned j = 0; j < n; j+=m){
            butter_prec<<<blocksPerGrid, threadsPerBlock>>>(d_x, d_twiddle, p, m, j, mid, i);
		}
    }

    int threadsPerBlockRev = 16;
    int blocksPerGridRev = (VECTOR_SIZE + threadsPerBlockRev - 1) / threadsPerBlockRev;
#ifndef TIME_TEST
    bit_reverse_cu<<<blocksPerGridRev, threadsPerBlockRev>>>(d_y, d_x, t);

    hipMemcpy(result, d_y, VECTOR_SIZE * sizeof(DATA_TYPE ), hipMemcpyDeviceToHost);
#endif

    return result;
}