#include "hip/hip_runtime.h"
#include "../include/ntt.h"
#include "../include/utils.h"
#include <inttypes.h>
using namespace std;

void cpyVec(DATA_TYPE* src, DATA_TYPE*dst, int length){
    for(int i=0; i<length; i++){
        dst[i] = src[i];
    }
}
__device__ void modulo_cu(int64_t base, int64_t m, DATA_TYPE &a){
	int64_t result = base % m;
	a = result >= 0? result : result + m;
}

__device__ void modExp_cu(DATA_TYPE base, unsigned exp, DATA_TYPE m, DATA_TYPE &a){

	DATA_TYPE result = 1;
	
	while(exp > 0){

		if(exp % 2){

			modulo_cu(result*base, m, result);

		}

		exp = exp >> 1;
		modulo_cu(base*base,m,base);
	}

    a = result;
}

__global__ void butter_prec(DATA_TYPE *y, DATA_TYPE *x, DATA_TYPE *d_twiddle, unsigned nn, unsigned s, unsigned shift, unsigned m, DATA_TYPE p) {
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    int mid = VECTOR_SIZE >> 1;
    if(t < mid){
        const int upper = (~(s - 1)) & t;
        DATA_TYPE w = d_twiddle[(upper >> shift) << shift];
        int q     = (s - 1) & t;
        DATA_TYPE a = y[q + (upper << 1)];
        DATA_TYPE b ;
        modulo_cu(y[q + (upper << 1) + s] * w, p, b);
        modulo_cu(a + b, p, x[q + upper]);
        modulo_cu(a - b, p, x[q + upper + s * m]);
    }
}

// GPU Mem pointer
DATA_TYPE* d_x ;
DATA_TYPE* d_y ;
DATA_TYPE* d_twiddle ;

void cudaPrepare(int n, DATA_TYPE p, DATA_TYPE g){
    hipDeviceProp_t myCuda;
    hipGetDeviceProperties(&myCuda, 0);
    cout<<"Cuda Name : "<<myCuda.name<<endl;
    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    cout<<"Cuda FreeMemory is" << freeMem<<" ; Total Memory is "<<totalMem<<endl;
    DATA_TYPE * twiddle = (DATA_TYPE*)malloc(n * sizeof(DATA_TYPE));
    DATA_TYPE w0 = modExp(g, (p - 1) / n, p);
    DATA_TYPE witer = 1;
    for(int i = 0; i < n; i++){
        twiddle[i] = witer;
        witer = modulo(witer * w0, p);
    }
    hipMalloc(&d_x, VECTOR_SIZE * sizeof(DATA_TYPE ));
    hipMalloc(&d_y, VECTOR_SIZE * sizeof(DATA_TYPE ));
    hipMalloc(&d_twiddle, VECTOR_SIZE * sizeof(DATA_TYPE ));
    hipMemcpy(d_twiddle, twiddle, VECTOR_SIZE * sizeof(DATA_TYPE ), hipMemcpyHostToDevice);
}

void hipFree(){
    hipFree(&d_twiddle);
    hipFree(&d_x);
    hipFree(&d_y);
}

DATA_TYPE* stockham_dit(DATA_TYPE* vec, DATA_TYPE n,  DATA_TYPE p,  DATA_TYPE g){

    unsigned t = (unsigned)log2(n);
    hipMemcpy(d_x, vec, VECTOR_SIZE * sizeof(DATA_TYPE ), hipMemcpyHostToDevice);
    int threadsPerBlock = 256;
    int blocksPerGrid = ((VECTOR_SIZE>>1) + threadsPerBlock - 1) / threadsPerBlock;

    DATA_TYPE * tmp = d_x;
    d_x = d_y;
    d_y = tmp;

    int s = n >> 1;
    int shift = t - 1;
    for(int nn = 2; nn <= n ; nn = nn << 1, s = s >> 1, shift = shift - 1){
        const int m = nn / 2;
        butter_prec<<<blocksPerGrid, threadsPerBlock>>>(d_y, d_x, d_twiddle, nn, s, shift, m, p);
        tmp = d_x;
        d_x = d_y;
        d_y = tmp;
    }   

    hipMemcpy(vec, d_y, VECTOR_SIZE * sizeof(DATA_TYPE ), hipMemcpyDeviceToHost);

    return vec;

}